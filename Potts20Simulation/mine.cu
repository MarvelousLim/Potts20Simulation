#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <time.h>

#define NNEIBORS 4 // number of nearest neighbors, is 4 for 2d lattice

// float precission
#define EPSILON 0.00001f

// check errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

bool cmpf(float x, float y) {
	return fabs(x - y) < EPSILON;
};

/*-----------------------------------------------------------------------------------------------------------
		Name agreement:

		s					array of all spins in spin-replica order
		SLF					spin lookup function for alone replica
		E					array of energies of replicas
		R					fixed population size (number of replicas)
		r					current replica number
		L					linear size of lattice
		N					fixed number of spins (N=L^2)
		j					current index inside alone replica
		q					Potts model parameter
		e					new spin value (char)
		n_i					neibors indexes inside alone replica
		n					neibors spin values
		replicaFamily		family (index of source replica after number of resamples); used to measure rho t
		rho_t				wtf value for checking equilibrium quality; A suitable condition is that rho_t << R
		energyOrder			ordering array, used during resampling step of algorithm
		MaxHistNumber		Maximum number of replicas to crease histogram from
		update              index of new replica to replace with

		To avoid confusion, lets describe the placement of the spin-replica array.
		This three-dimensional structure (L_x * L_y * R) lies like one-dimensional array,
		first goes, one by one, strings of first replica, then second etc. Here we calculate
		everything inside one replica, adding r factor later

		Also, when its about generation random numbers, we use R threads, one for each replica

-------------------------------------------------------------------------------------------------------------*/

struct neibors_indexes {
	int up;
	int right;
	int down;
	int left;
};

__host__ __device__ struct neibors_indexes SLF(int j, int L, int N) {
	struct neibors_indexes result;
	result.up = (j - L + N) % N; // N member is for positivity
	result.right = (j + 1) % L + L * (j / L);
	result.down = (j + L) % N;
	result.left = (j - 1 + L) % L + L * (j / L); // L member is for positivity
	return result;
}

struct neibors {
	char up;
	char right;
	char down;
	char left;
};

struct energy_parts {
	int Ising;
	int Blume;
};

__device__ struct neibors get_neibors_values(char* s, struct neibors_indexes n_i, int replica_shift) {
	return {
		s[n_i.up + replica_shift],
		s[n_i.right + replica_shift],
		s[n_i.down + replica_shift],
		s[n_i.left + replica_shift]
	};
}

__host__ __device__ struct energy_parts localEnergyParts(char currentSpin, struct neibors n) {
	// Computes energy of spin i with neighbors a, b, c, d 
	// 2 * D due to following halfening
	return {
		- (currentSpin * n.up)
		- (currentSpin * n.right)
		- (currentSpin * n.down)
		- (currentSpin * n.left)
		, (currentSpin * currentSpin)
	};
}

__device__ struct energy_parts addEnergyParts(struct energy_parts A, struct energy_parts B) {
	return { A.Ising + B.Ising, A.Blume + B.Blume };
}

__device__ struct energy_parts subEnergyParts(struct energy_parts A, struct energy_parts B) {
	return { A.Ising - B.Ising, A.Blume - B.Blume };
}

__device__ struct energy_parts deltaLocalEnergyParts(char currentSpin, char suggestedSpin, struct neibors n) { // Delta of local energy while i -> e switch
	struct energy_parts suggestedEnergyParts = localEnergyParts(suggestedSpin, n);
	struct energy_parts currentEnergyParts = localEnergyParts(currentSpin, n);
	return subEnergyParts(suggestedEnergyParts, currentEnergyParts);
}

__device__ struct energy_parts calcEnergyParts(char* s, float* E, int L, int N, float D, int r) {
	struct energy_parts sum = { 0, 0 };
	for (int j = 0; j < N; j++) {
		// do not forget double joint summarization!
		int replica_shift = r * N;
		char i = s[j + replica_shift]; // current spin value
		struct neibors_indexes n_i = SLF(j, L, N);
		struct neibors n = get_neibors_values(s, n_i, replica_shift); // we look into r replica and j spin
		struct energy_parts tmp = localEnergyParts(i, n);
		sum = addEnergyParts(sum, tmp);
	}
	return sum;
}

__device__ float calcEnergyFromParts(struct energy_parts energyParts, float D) {
	return (energyParts.Ising / 2) + (D * energyParts.Blume); // div 2 because of double joint summarization
}

__global__ void deviceEnergy(char* s, float* E, int L, int N, float D) {
	int r = threadIdx.x + blockIdx.x * blockDim.x;
	struct energy_parts sum = calcEnergyParts(s, E, L, N, D, r);
	E[r] = calcEnergyFromParts(sum, D); 
}

// hardcoded spin suggestion
__device__ char suggestSpin(hiprandStatePhilox4_32_10_t* state, int r) {
	return hiprand(&state[r]) % 3 - 1;
}

__device__ char suggestSpinSwap(hiprandStatePhilox4_32_10_t* state, int r, char currentSpin) {
	return (currentSpin + 2 + (hiprand(&state[r]) % 2)) % 3 - 1; // little trick
}

__global__ void equilibrate(hiprandStatePhilox4_32_10_t* state, char* s, float* E, int L, int N, int R, int q, int nSteps, float U, float D, bool heat){//, int* acceptance_number) {
	/*---------------------------------------------------------------------------------------------
		Main Microcanonical Monte Carlo loop.  Performs update sweeps on each replica in the
		population;
		There, one could change calcEnergyParts for system of carrying arrays of energy parts,
		but:
			1. This is not the bottleneck (which is for loop over N * nSteps (could be halved?))
			2. ...
			3. Lazy to assign memmory for it
	---------------------------------------------------------------------------------------------*/

	int r = threadIdx.x + blockIdx.x * blockDim.x;
	int replica_shift = r * N;

	struct energy_parts baseEnergyParts = calcEnergyParts(s, E, L, N, D, r);

	for (int k = 0; k < N * nSteps; k++) {
		int j = hiprand(&state[r]) % N;
		char currentSpin = s[j + replica_shift];
		char suggestedSpin = suggestSpinSwap(state, r, currentSpin);
		//char suggestedSpin = curand(&state[r]) % 3 - 1;
		struct neibors_indexes n_i = SLF(j, L, N);
		struct neibors n = get_neibors_values(s, n_i, replica_shift);
		struct energy_parts deltaEnergyParts = deltaLocalEnergyParts(currentSpin, suggestedSpin, n);
		struct energy_parts suggestedEnergyParts = addEnergyParts(baseEnergyParts, deltaEnergyParts);
		float suggestedEnergy = calcEnergyFromParts(suggestedEnergyParts, D);
		
		if (( !heat && (suggestedEnergy + EPSILON < U) ) || (heat && (suggestedEnergy - EPSILON > U) )) {
			baseEnergyParts = suggestedEnergyParts;
			E[r] = suggestedEnergy;
			s[j + replica_shift] = suggestedSpin;
		}
	}
}

void CalcPrintAvgE(FILE* efile, float* E, int R, float U) {
	float avg = 0.0;
	for (int i = 0; i < R; i++) {
		avg += E[i];
	}
	avg /= R;
	fprintf(efile, "%f %f\n", U, avg);
	printf("E: %f\n", avg);
}

void CalculateRhoT(const int* replicaFamily, FILE* ptfile, int R, float U) {
	// histogram of family sizes
	int* famHist = (int*)calloc(R, sizeof(int));
	for (int i = 0; i < R; i++) {
		famHist[replicaFamily[i]]++;
	}
	double sum = 0;
	for (int i = 0; i < R; i++) {
		sum += famHist[i] * famHist[i];
	}
	sum /= R;
	fprintf(ptfile, "%f %f\n", U, sum);
	sum /= R;
	printf("RhoT:\t%f\n", sum);
	free(famHist);
}

__global__ void initializePopulation(hiprandStatePhilox4_32_10_t* state, char* s, int N, int q) {
	/*---------------------------------------------------------------------------------------------
		Initializes population on gpu(!) by randomly assigning each spin a value from 0 to q-1
	----------------------------------------------------------------------------------------------*/
	int r = threadIdx.x + blockIdx.x * blockDim.x;
	for (int k = 0; k < N; k++) {
		int arrayIndex = r * N + k;
		char spin = suggestSpin(state, r);
		s[arrayIndex] = spin;
	}
}

void Swap(int* A, int i, int j) {
	int temp = A[i];
	A[i] = A[j];
	A[j] = temp;
}

void quicksort(float* E, int* O, int left, int right, int direction) {
	int Min = (left + right) / 2;
	int i = left;
	int j = right;
	double pivot = direction * E[O[Min]];

	while (left < j || i < right)
	{
		while (direction * E[O[i]] > pivot)
			i++;
		while (direction * E[O[j]] < pivot)
			j--;

		if (i <= j) {
			Swap(O, i, j);
			i++;
			j--;
		}
		else {
			if (left < j)
				quicksort(E, O, left, j, direction);
			if (i < right)
				quicksort(E, O, i, right, direction);
			return;
		}
	}
}

int resample(float* E, int* O, int* update, int* replicaFamily, int R, float* U, FILE* e2file, FILE* Xfile, bool heat) {
	//std::sort(O, O + R, [&E](int a, int b) {return E[a] > E[b]; }); // greater sign for descending order
	quicksort(E, O, 0, R - 1, 1 - 2 * heat); //Sorts O by energy

	int nCull = 0;
	fprintf(e2file, "%f %f\n", U, E[O[0]]);

	//update energy seiling to the highest available energy
	float U_old = *U;
	float U_new;
	
	for (int i = 0; i < R; i++) {
		U_new = E[O[i]];
		if ((!heat && U_new < U_old - EPSILON) || (heat && U_new > U_old + EPSILON)) {
			*U = U_new;
			break;
		}
	}

	if (fabs(*U - U_old) < EPSILON) {
		return 1; // out of replicas
	}

	while ((!heat && E[O[nCull]] >= *U - EPSILON) || (heat && E[O[nCull]] <= *U + EPSILON)) {
		nCull++;
		if (nCull == R) {
			break;
		}
	}
	// culling fraction
	double X = nCull;
	X /= R;
	fprintf(Xfile, "%f %f\n", *U, X);
	printf("Culling fraction:\t%f\n", X);
	for (int i = 0; i < R; i++)
		update[i] = i;
	if (nCull < R) {
		for (int i = 0; i < nCull; i++) {
			// random selection of unculled replica
			int r = (rand() % (R - nCull)) + nCull; // different random number generator for resampling
			update[O[i]] = O[r];
			replicaFamily[O[i]] = replicaFamily[O[r]];
		}
	}

	return 0;
}

__global__ void updateReplicas(char* s, float* E, int* update, int N) {
	/*---------------------------------------------------------------------------------------------
		Updates the population after the resampling step (done on cpu) by replacing indicated
		replicas by the proper other replica
	-----------------------------------------------------------------------------------------------*/
	int r = threadIdx.x + blockIdx.x * blockDim.x;
	int replica_shift = r * N;
	int source_r = update[r];
	int source_replica_shift = source_r * N;
	if (source_r != r) {
		for (int j = 0; j < N; j++) {
			s[j + replica_shift] = s[j + source_replica_shift];
		}
		E[r] = E[update[r]];
	}
}

__global__ void setup_kernel(hiprandStatePhilox4_32_10_t* state, int seed)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	/* Each thread gets same seed, a different sequence
	   number, no offset */
	hiprand_init(seed, id, 0, state + id);
}

int main(int argc, char* argv[]) {

	// Parameters:

	int run_number = atoi(argv[1]);	// A number to label this run of the algorithm, used for data keeping purposes, also, a seed
	int seed = run_number;
	//int grid_width = atoi(argv[2]);	// should not be more than 256 due to MTGP32 limits
	int L = atoi(argv[2]);	// Lattice size
	int N = L * L;
	//int R = grid_width * BLOCKS;	// Population size
	int BLOCKS = atoi(argv[3]);
	int THREADS = atoi(argv[4]);
	int nSteps = atoi(argv[5]);

	int R = BLOCKS * THREADS;

	// q parameter for potts model, each spin variable can take on values 0 - q-1
	// strictly hardcoded
	int q = 3;

	//Blume-Capel model parameter
	float D = atof(argv[6]);
	bool heat = atoi(argv[7]); // 0 if cooling (default) and 1 if heating


	// initializing files to write in
	const char* heating = heat ? "Heating" : "";

	printf("running 2DBlume%s_q%d_D%f_N%d_R%d_nSteps%d_run%de.txt\n", heating, q, D, N, R, nSteps, run_number);

	char s[100];
	sprintf(s, "datasets//2DBlume%s_q%d_D%f_N%d_R%d_nSteps%d_run%de.txt", heating, q, D, N, R, nSteps, run_number);
	FILE* efile = fopen(s, "w");	// average energy
	sprintf(s, "datasets//2DBlume%s_q%d_D%f_N%d_R%d_nSteps%d_run%de2.txt", heating, q, D, N, R, nSteps, run_number);
	FILE* e2file = fopen(s, "w");	// surface (culled) energy
	sprintf(s, "datasets//2DBlume%s_q%d_D%f_N%d_R%d_nSteps%d_run%dX.txt", heating, q, D, N, R, nSteps, run_number);
	FILE* Xfile = fopen(s, "w");	// culling fraction
	sprintf(s, "datasets//2DBlume%s_q%d_D%f_N%d_R%d_nSteps%d_run%dpt.txt", heating, q, D, N, R, nSteps, run_number);
	FILE* ptfile = fopen(s, "w");	// rho t
	sprintf(s, "datasets//2DBlume%s_q%d_D%f_N%d_R%d_nSteps%d_run%dn.txt", heating, q, D, N, R, nSteps, run_number);
	FILE* nfile = fopen(s, "w");	// number of sweeps
	sprintf(s, "datasets//2DBlume%s_q%d_D%f_N%d_R%d_nSteps%d_run%dch.txt", heating, q, D, N, R, nSteps, run_number);
	FILE* chfile = fopen(s, "w");	// cluster size histogram


	size_t fullLatticeByteSize = R * N * sizeof(char);

	// Allocate space on host
	float* hostE = (float*)malloc(R * sizeof(float));
	int* hostUpdate = (int*)malloc(R * sizeof(int));
	int* replicaFamily = (int*)malloc(R * sizeof(int));
	int* energyOrder = (int*)malloc(R * sizeof(int));
	for (int i = 0; i < R; i++) {
		energyOrder[i] = i;
		replicaFamily[i] = i;
	}

	// Allocate memory on device
	char* deviceSpin; // s, d_s
	float* deviceE;
	int* deviceUpdate;
	gpuErrchk( hipMalloc((void**)&deviceSpin, fullLatticeByteSize) );
	gpuErrchk( hipMalloc((void**)&deviceE, R * sizeof(float)) );
	gpuErrchk( hipMalloc((void**)&deviceUpdate, R * sizeof(int)) );

	// Allocate memory for histogram calculation
	/*
	int* hostClusterSizeArray = (int*)malloc(N * sizeof(int));
	bool* deviceVisited;
	int* deviceClusterSizeArray;
	int* deviceStack;
	gpuErrchk( cudaMalloc((void**)&deviceVisited, N * R * sizeof(bool)) );
	gpuErrchk( cudaMalloc((void**)&deviceClusterSizeArray, N * sizeof(int)) );
	gpuErrchk( cudaMalloc((void**)&deviceStack, N * R * sizeof(int)) );
	*/

	// Init Philox
	hiprandStatePhilox4_32_10_t* devStates;
	gpuErrchk( hipMalloc((void**)&devStates, R * sizeof(hiprandStatePhilox4_32_10_t)) );
	setup_kernel <<< BLOCKS, THREADS >>> (devStates, seed);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// Init std random generator for little host part
	srand(seed);

	// Actually working part
	initializePopulation <<< BLOCKS, THREADS >>> (devStates, deviceSpin, N, q);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	hipMemset(deviceE, 0, R * sizeof(int));

	/*
	//init testing values 
	deviceEnergy <<< BLOCKS, THREADS >>> (deviceSpin, deviceE, L, N, D);
	gpuErrchk(cudaPeekAtLastError());
	gpuErrchk(cudaDeviceSynchronize());
	gpuErrchk( cudaMemcpy(hostE, deviceE, R * sizeof(int), cudaMemcpyDeviceToHost) );
	int* device_acceptance_number;
	gpuErrchk( cudaMalloc((void**)&device_acceptance_number, sizeof(int)) );
	char* hostSpin = (char*)malloc(N * sizeof(char)); // test shit
	int host_acceptance_number = 0;
	*/

	float upper_energy = N * D + 2 * N;
	float lower_energy = - N * D - 2 * N;
	float U = (heat ? lower_energy : upper_energy);	// U is energy ceiling

	//CalcPrintAvgE(efile, hostE, R, U);

	
	while ((U >= lower_energy && !heat) || (U <= upper_energy && heat)) {
		fprintf(nfile, "%f %d\n", U, nSteps);
		printf("U:\t%f out of %d; nSteps: %d;\n", U, -2 * N, nSteps);

		// Perform monte carlo sweeps on gpu
		//clock_t begin = clock();

		//cudaMemset(device_acceptance_number, 0, sizeof(int));

		equilibrate <<< BLOCKS, THREADS >>> (devStates, deviceSpin, deviceE, L, N, R, q, nSteps, U, D, heat);//, device_acceptance_number);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		/*
		gpuErrchk(cudaMemcpy(&host_acceptance_number, device_acceptance_number, sizeof(int), cudaMemcpyDeviceToHost));
		printf("acceptance_number: %i\nacceptance_ratio: %02f \n", host_acceptance_number, 100.0 * host_acceptance_number / (N * R * nSteps) );

		clock_t end = clock();
		double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		printf("Time: %f seconds\n", time_spent);
		*/
		//cudaDeviceSynchronize();

		// Create disordered cluster size histogram in particular energy range
		//if (U <= -1.5 * N)
		//	makeClusterHistogram(deviceSpin, deviceE, N, L, BLOCKS, THREADS, U, chfile, deviceVisited, deviceClusterSizeArray, deviceStack, hostClusterSizeArray);

		//deviceEnergy <<< BLOCKS, THREADS >>> (deviceSpin, deviceE, L, N, D);
		//gpuErrchk(cudaPeekAtLastError());
		//gpuErrchk(cudaDeviceSynchronize());
		
		gpuErrchk( hipMemcpy(hostE, deviceE, R * sizeof(int), hipMemcpyDeviceToHost) );

		/*
		printf("E: ");
		for (int i = 0; i < 10; i++) {
			printf("%f ", hostE[i]);
		}
		printf("\n");
		*/

		// record average energy and rho t
		CalcPrintAvgE(efile, hostE, R, U);
		CalculateRhoT(replicaFamily, ptfile, R, U);
		// perform resampling step on cpu
		// also lowers energy seiling U
		
		int error = resample(hostE, energyOrder, hostUpdate, replicaFamily, R, &U, e2file, Xfile, heat);
		if (error)
		{
			printf("Process ended with zero replicas\n");
			break;
		}
		// copy list of replicas to update back to gpu
		gpuErrchk( hipMemcpy(deviceUpdate, hostUpdate, R * sizeof(int), hipMemcpyHostToDevice) );
		updateReplicas <<< BLOCKS, THREADS >>> (deviceSpin, deviceE, deviceUpdate, N);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
		printf("\n");
	}
	

	// Free memory and close files
	hipFree(devStates);
	hipFree(deviceSpin);
	hipFree(deviceE);
	hipFree(deviceUpdate);
	//cudaFree(deviceClusterSizeArray);
	//cudaFree(deviceStack);
	//cudaFree(deviceVisited);
	//cudaFree(device_acceptance_number);

	free(hostE);
	free(hostUpdate);
	free(replicaFamily);
	free(energyOrder);
	//free(hostClusterSizeArray);
	//free(hostSpin);

	fclose(efile);
	fclose(e2file);
	fclose(Xfile);
	fclose(ptfile);
	fclose(nfile);
	fclose(chfile);

	// End
	return 0;
}


