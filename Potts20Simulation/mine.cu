#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <queue>
#include <vector>
#include <map>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#define NNEIBORS 4 // number of nearest neighbors, is 4 for 2d lattice
#define BLOCKS 200
//#define MAX_THREADS 256 // hiprand property
//#define MAX_NGENERATORS BLOCKS * THREADS

/*-----------------------------------------------------------------------------------------------------------
		Name agreement:

		s					array of all spins in spin-replica order
		SLF					spin lookup function for alone replica
		E					array of energies of replicas
		R					fixed population size (number of replicas)
		r					current replica number
		L					linear size of lattice
		N					fixed number of spins (N=L^2)
		j					current index inside alone replica
		q					Potts model parameter
		e					new spin value (char)
		n_i					neibors indexes inside alone replica
		n					neibors spin values
		replicaFamily		family (index of source replica after number of resamples); used to measure rho t
		rho_t				wtf value for checking equilibrium quality; A suitable condition is that rho_t << R
		energyOrder			ordering array, used during resampling step of algorithm
		MaxHistNumber		Maximum number of replicas to crease histogram from
		update              index of new replica to replace with

		To avoid confusion, lets describe the placement of the spin-replica array.
		This three-dimensional structure (L_x * L_y * R) lies like one-dimensional array,
		first goes, one by one, strings of first replica, then second etc. Here we calculate
		everything inside one replica, adding r factor later

		Also, when its about generation random numbers, we use R threads, one for each replica

-------------------------------------------------------------------------------------------------------------*/

// more or less host function

struct neibors_indexes {
	int up;
	int down;
	int left;
	int right;
};

__host__ __device__ struct neibors_indexes SLF(int j, int L, int N) {
	struct neibors_indexes result;
	result.right = (j + 1) % L + L * (j / L);
	result.left = (j - 1 + L) % L + L * (j / L); // L member is for positivity
	result.down = (j + L) % N;
	result.up = (j - L + N) % N; // N member is for positivity
	return result;
}

struct neibors {
	char up;
	char down;
	char left;
	char right;
};

__host__ __device__ struct neibors get_neibors_values(char* s, struct neibors_indexes n_i, int replica_shift) {
	return { s[n_i.up + replica_shift], s[n_i.down + replica_shift], s[n_i.left + replica_shift], s[n_i.right + replica_shift] };
}


__host__ __device__ int LocalE(char currentSpin, struct neibors n) { 	// Computes energy of spin i with neighbors a, b, c, d 
	return -(currentSpin == n.up) - (currentSpin == n.down) - (currentSpin == n.left) - (currentSpin == n.right);
}
__host__ __device__ int DeltaE(char currentSpin, char suggestedSpin, struct neibors n) { // Delta of local energy while i -> e switch
	return LocalE(suggestedSpin, n) - LocalE(currentSpin, n);
}


void hostEnergy(char* s, thrust::host_vector<int>& E, int R, int L, int N) {
	for (int r = 0; r < R; r++) {
		int sum = 0;
		for (int j = 0; j < N; j++) {
			// 0.5 by doubling the summarize (stupid)
			int replica_shift = r * N;
			char i = s[j + replica_shift]; // current spin value
			struct neibors_indexes n_i = SLF(j, L, N);
			struct neibors n = get_neibors_values(s, n_i, replica_shift); // we look into r replica and j spin
			sum += LocalE(i, n);
		}
		E[r] = sum / 2;
	}
}

void CalcPrintAvgE(std::ofstream& efile, const thrust::host_vector<int>& E, int U, int R) {
	float avg = 0.0;
	for (int i = 0; i < R; i++)
		avg += E[i];
	avg /= R;
	efile << U << " " << avg << std::endl;
}

// Part of disordered cluster histogram algorithm
int BFS(char* s, int start, std::vector<bool>& visited, int L, int N, int replica_shift) {

	std::queue<int> BFS_queue;
	BFS_queue.push(start);
	visited[start] = 1;
	int clusterSize = 0;
	char spinValue = s[start + replica_shift];

	while (!BFS_queue.empty()) {
		int currentIndex = BFS_queue.front();
		BFS_queue.pop();
		clusterSize++;
		struct neibors_indexes n = SLF(currentIndex, L, N);
		std::vector<int> possibleIndexes = { n.up, n.down, n.left, n.right };
		for (int newIndex : possibleIndexes)
			if ((s[newIndex + replica_shift] == spinValue) && (!visited[newIndex])) {
				BFS_queue.push(newIndex);
				visited[newIndex] = 1;
			}
	}
	return clusterSize;
}

void makeClusterHistogram(char* s, thrust::host_vector<int>& E, int q, int N, int L, int R, int MaxHistNumber, int U, std::ofstream& chfile) {
	/*------------------------------------------------------------------------------------------------
		Disordered Cluster Histogram Algorithm
		Steps to procedure:
			- Identify replicas with energy one less than the energy ceiling
			x Determine the Majority (ordered) cluster
			x Identify clusters of spins not in the majority phase. For low
				enough energies, these should be isolated clusters.
			+ I decided to calculate all cluster, because why not. Can always kill em on postcalc
			- compile histogram of cluster sizes
	-------------------------------------------------------------------------------------------------*/
	std::map<int, int> clusterSizeFreqMap; // should be safe to assume the default value as 0 (?)
	int HistNumber = 0;
	for (int r = 0; r < R; r++) {
		std::vector<bool> visited(N); // default-false for every replica
		int replica_shift = r * N;
		if (E[r] == U - 1 && HistNumber < MaxHistNumber) {
			HistNumber++;
			// Go sequentially through lattice and assign spins to clusters
			for (int i = 0; i < N; i++) {
				if (!visited[i]) {
					int currentClusteSize = BFS(s, i, visited, L, N, replica_shift);
					clusterSizeFreqMap[currentClusteSize]++;
				}
			}
		}
		else if (HistNumber >= MaxHistNumber)
			break;
	}
	// write results to output files
	if (HistNumber > 0) {
		chfile << HistNumber << " ";
		for (auto& item : clusterSizeFreqMap) {
			int size = item.first;
			int freq = item.second;
			chfile << size << " " << freq << std::endl;
		}
	}
}

void CalculateRhoT(const thrust::host_vector<int>& replicaFamily, int R, std::ofstream& ptfile, int U) {
	// histogram of family sizes
	std::vector<int> famHist(R);
	for (int i = 0; i < R; i++) {
		famHist[replicaFamily[i]]++;
	}
	float sum = 0;
	for (int i = 0; i < R; i++) {
		sum += famHist[i] * famHist[i];
	}
	sum /= R;
	ptfile << U << " " << sum << std::endl;
}


// device functions

__global__ void initializePopulation(hiprandStateMtgp32* state, char* s, int N, int q, int R) {
	/*---------------------------------------------------------------------------------------------
		Initializes population on gpu(!) by randomly assigning each spin a value from 0 to q-1
	----------------------------------------------------------------------------------------------*/
	int r = threadIdx.x + blockIdx.x * blockDim.x;
	for (int k = 0; k < N; k++) {
		int arrayIndex = r * N + k;
		char spin = hiprand(&state[blockIdx.x]) % q;
		s[arrayIndex] = spin;
	}
}

__global__ void equilibrate(hiprandStateMtgp32* state, char* s, int* E, int L, int N, int R, int q, int nSteps, int U) {
	/*---------------------------------------------------------------------------------------------
		Main Microcanonical Monte Carlo loop.  Performs update sweeps on each replica in the
		population;
	---------------------------------------------------------------------------------------------*/

	int r = threadIdx.x + blockIdx.x * blockDim.x;
	int replica_shift = r * N;
	for (int k = 0; k < N * nSteps; k++) {
		int j = hiprand(&state[blockIdx.x]) % N;
		char currentSpin = s[j + replica_shift];
		char suggestedSpin = hiprand(&state[blockIdx.x]) % q;
		struct neibors_indexes n_i = SLF(j, L, N);
		struct neibors n = get_neibors_values(s, n_i, replica_shift);
		int dE = DeltaE(currentSpin, suggestedSpin, n);
		if (E[r] + dE < U) {
			E[r] = E[r] + dE;
			s[j + replica_shift] = suggestedSpin;
		}
	}
}

void resample(const thrust::host_vector<int>& E, thrust::host_vector<int>& O, thrust::host_vector<int>& update,
	thrust::host_vector<int>& replicaFamily, int R, int U, std::ofstream& e2file, std::ofstream& Xfile) {
	//quicksort(E, O, 0, R - 1); // Sorts O by energy
	thrust::sort(O.begin(), O.end(), [&E](int a, int b) {return E[a] > E[b]; }); // greater sign for descending order
	int nCull = 0;
	e2file << U << " " << E[O[0]] << std::endl;
	while (E[O[nCull]] == U - 1) {
		nCull++;
		if (nCull == R) {
			break;
		}
	}
	// culling fraction
	double X = (double)nCull / R;
	Xfile << U << " " << X << std::endl;
	std::cout << "Culling fraction:\t" << X << std::endl;
	for (int i = 0; i < R; i++)
		update[i] = i;
	if (nCull < R) {
		for (int i = 0; i < nCull; i++) {
			// random selection of unculled replica
			int r = (std::rand() % (R - nCull)) + nCull; // different random number generator for
			update[O[i]] = O[r];
			replicaFamily[O[i]] = replicaFamily[O[r]];
		}
	}
}

__global__ void updateReplicas(char* s, int* E, int* update, int N, int R) {
	/*---------------------------------------------------------------------------------------------
		Updates the population after the resampling step (done on cpu) by replacing indicated
		replicas by the proper other replica
	-----------------------------------------------------------------------------------------------*/
	int r = threadIdx.x + blockIdx.x * blockDim.x;
	int replica_shift = r * N;
	int source_r = update[r];
	int source_replica_shift = source_r * N;
	if (source_r != r) {
		for (int j = 0; j < N; j++) {
			s[j + replica_shift] = s[j + source_replica_shift];
		}
		E[r] = E[update[r]];
	}
}

template<class T>
void PrintVector(const T& v, std::string prefix) {
	std::cout << prefix;
	for (auto& item : v)
		std::cout << item << " ";
	std::cout << std::endl;
}

void PrintArray(char* s, std::string prefix, int L, int R) {
	std::cout << prefix << std::endl;
	int N = L * L;
	for (int r = 0; r < R; r++) {
		std::cout << "replica " << r << std::endl;
		for (int i = 0; i < L; i++) {
			for (int j = 0; j < L; j++) {
				std::cout << (int)s[j + i * L + r * N] << "\t";
			}
			std::cout << std::endl;
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

int main(int argc, char* argv[]) {
	// Parameters:
	int nSteps = 1;
	int q = 20;	// q parameter for potts model, each spin variable can take on values 0 - q-1
	int U = 1;	// U is energy ceiling

	// random number generation
	hiprandStateMtgp32* devMTGPStates;
	mtgp32_kernel_params* devKernelParams;

	int run_number = atoi(argv[1]);	// A number to label this run of the algorithm, used for data keeping purposes, also, a seed
	int seed = run_number;
	int grid_width = atoi(argv[2]);	// should not be more than 256 due to MTGP32 limits
	int L = atoi(argv[3]);	//Lattice size.  Total number of spins is N=L^2
	int N = L * L;
	int R = grid_width * BLOCKS;	// Population size

	// initializing files to write in
	std::stringstream s;
	s << "datasets//L" << L << "_R" << R << "_run" << run_number;
	std::string S = s.str();
	std::ofstream efile(S + "e.txt", std::ofstream::trunc);	// average energy
	std::ofstream e2file(S + "e2.txt", std::ofstream::trunc);	// surface (culled) energy
	std::ofstream Xfile(S + "X.txt", std::ofstream::trunc);	// culling fraction
	std::ofstream ptfile(S + "pt.txt", std::ofstream::trunc);	// rho t
	std::ofstream nfile(S + "n.txt", std::ofstream::trunc);	// number of sweeps
	std::ofstream chfile(S + "ch.txt", std::ofstream::trunc);	// cluster size histogram


	size_t fullLatticeByteSize = R * N * sizeof(char);

	// Allocate space on host 
	char* hostSpin = (char*)malloc(fullLatticeByteSize); // s, h_s
	thrust::host_vector<int> hostE(R);
	thrust::host_vector<int> replicaFamily(R);
	thrust::host_vector<int> energyOrder(R);
	thrust::host_vector<int> hostUpdate(R);
	for (int i = 0; i < R; i++) {
		energyOrder[i] = i;
		replicaFamily[i] = i;
	}
	// Allocate memory on device
	char* deviceSpin; // s, d_s
	thrust::device_vector<int> deviceE(R);
	thrust::device_vector<int> deviceUpdate(R);
	int* deviceEPointer = thrust::raw_pointer_cast(deviceE.data()); // assume, that pointers does not changes since there are no reallocations
	int* deviceUpdatePointer = thrust::raw_pointer_cast(deviceUpdate.data());
	hipMalloc((void**)&deviceSpin, fullLatticeByteSize);

	// Init MTGP32
	hipMalloc((void**)&devMTGPStates, BLOCKS * sizeof(hiprandStateMtgp32));
	hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params));
	hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams);
	hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, BLOCKS, seed);

	// Init std random generator for little host part
	std::srand(seed);

	// Actually working part
	initializePopulation << <BLOCKS, grid_width >> > (devMTGPStates, deviceSpin, N, q, R);
	hipMemcpy(hostSpin, deviceSpin, fullLatticeByteSize, hipMemcpyDeviceToHost);
	//	PrintArray(hostSpin, "s", L, R); // debug
	hostEnergy(hostSpin, hostE, R, L, N);
	//	PrintVector(hostE, "E "); // debug
	deviceE = hostE;

	int loop = 0;
	while (U > -2 * N) {
		loop++;
		// Adjust the sweep schedule
		// Most sweeps are performed in the region when simulation is most difficult
		if (U < -(3 * N / 2))
			nSteps = 10;
		else if (U < -N / 2)
			nSteps = 30;
		else // (U >= -N / 2)
			nSteps = 2;

		nfile << U << " " << nSteps << std::endl;
		std::cout << "U:\t" << U << " out of " << -2 * N << "; nSteps: " << nSteps << ";" << std::endl;
		// Perform monte carlo sweeps on gpu
		equilibrate << <BLOCKS, grid_width >> > (devMTGPStates, deviceSpin, deviceEPointer, L, N, R, q, nSteps, U);
		//		PrintArray(hostSpin, "s", L, R); // debug
		hostE = deviceE;
		//		PrintVector(hostE, "E "); // debug

		int numHist = 100000;
		// Create disordered cluster size histogram in particular energy range
		if (U <= -1.5 * N) {
			// Copy energy and spin configuration back to host
			hipMemcpy(hostSpin, deviceSpin, fullLatticeByteSize, hipMemcpyDeviceToHost);
			makeClusterHistogram(hostSpin, hostE, q, N, L, R, numHist, U, chfile);
		}
		// record average energy and rho t
		CalcPrintAvgE(efile, hostE, U, R);
		CalculateRhoT(replicaFamily, R, ptfile, U);
		// perform resampling step on cpu
		resample(hostE, energyOrder, hostUpdate, replicaFamily, R, U, e2file, Xfile);
		U--;
		//		PrintVector(energyOrder, "O "); // debug
		//		PrintVector(hostUpdate, "update "); // debug
				// copy list of replicas to update back to gpu
		deviceUpdate = hostUpdate;
		updateReplicas << <BLOCKS, grid_width >> > (deviceSpin, deviceEPointer, deviceUpdatePointer, N, R);
		//		hipMemcpy(hostSpin, deviceSpin, fullLatticeByteSize, hipMemcpyDeviceToHost); // debug
		//		PrintArray(hostSpin, "s_updated", L, R); // debug
		//		PrintVector(replicaFamily, "replicaFamily "); // debug
	}
	CalcPrintAvgE(efile, hostE, U, R);

	// Free memory and close files
	hipFree(devMTGPStates);
	hipFree(devKernelParams);
	hipFree(deviceSpin);
	free(hostSpin);
	efile.close();
	e2file.close();
	Xfile.close();
	ptfile.close();
	nfile.close();
	chfile.close();

	// End
	return 0;
}
