#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
//#include <mt19937.h>

#define NNEIBORS 4 // number of nearest neighbors, is 4 for 2d lattice
#define BLOCKS 64 //64
#define THREADS 256 // 64
#define NGENERATORS BLOCKS * THREADS
#define LatticeType float

// Computes energy delta from flipping spin at site i to value e with neighbors a,b,c,d 
template <class T>
__global__	int del(T i, T a, T b, T c, T d, T e) {
	return (i == a) + (i == b) + (i == c) + (i == d) - (e == a) - (e == b) - (e == c) - (e == d);
}

__global__ void initLattice(hiprandStateMtgp32* state, LatticeType* devR, int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	float r;
	if (tid < NGENERATORS)
	{
		for (int i = tid; i < N; i += NGENERATORS)
		{
			devR[i] = hiprand(&state[blockIdx.x]);
		}
	}
}


int reset_timers(clock_t* previous, clock_t* current) // not pure code, but unimportante
{
	*previous = *current;
	*current = clock();
	return *current - *previous;
}

int main(int argc, char* argv[]) {
	// Parameters:
	clock_t begin = clock(), previous = begin, current = begin;
	// random number generation
	hiprandStateMtgp32* devMTGPStates;
	mtgp32_kernel_params* devKernelParams;
	LatticeType* R, * devR;
	int N = 10, seed = 0;

	// Allocate space for lattice on host 
	R = (LatticeType*)calloc(N, sizeof(LatticeType));

	// Allocate space for results on device 
	hipMalloc((void**)&devR, N * sizeof(LatticeType));

	// Init MTRG
	hipMalloc((void**)&devMTGPStates, BLOCKS * sizeof(hiprandStateMtgp32));
	hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params));
	hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams);
	hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, BLOCKS, seed);

	std::cout << "Initialization took: " << reset_timers(&previous, &current) << " clocks;\n";
	//actually working part


	initLattice <<< BLOCKS, THREADS >>> (devMTGPStates, devR, N);
	hipMemcpy(R, devR, N * sizeof(LatticeType), hipMemcpyDeviceToHost);

	//end of acctually working part

	for (int i = 0; i < N; i++)
		std::cout << R[i] << " ";
	std::cout << "\n" << "Whole programm took: " << reset_timers(&previous, &begin) << " clocks;\n";

	free(R);
	hipFree(devMTGPStates);
	hipFree(devKernelParams);
	hipFree(devR);
	return 0;
}
